#include "hip/hip_runtime.h"
// -*- mode: C -*-
/* ==========================================================================================
                                 GPU_2DIsing.cu

Implementation of the 2D Ising model in CUDA. Each CUDA thread simulates an independent 
instance of the 2D Ising model in parallel with an independent random number sequence. Draws
heavily from the work of Weigel et al, [J. Phys.: Conf. Ser.921 012017 (2017)] but used here
for gathering rare event statistics on nucleation during magnetisation reversal. 
 ===========================================================================================*/
// D. Quigley. Univeristy of Warwick

// TODO
// 1. sweep counter probably needs to be a long and not an int
// 2. clustering on CPU asynchronously with GPU ?
// 3. write magnetisation to binary file ?

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>  
#include <float.h>
#include <stdbool.h>

extern "C" {
  #include "mc_cpu.h"
  #include "io.h"
  #include "parser.h"
}

#include "mc_gpu.h"
#include "gpu_tools.h"


const bool run_gpu = true;      // Run using GPU
const bool run_cpu = false;     // Run using CPU



int main (int argc, char *argv[]) {

/*=================================
    Parse Commandline arguments
  =================================*/ 
  // set default arguments
  //  currently set the same as command in readme
  default_args = (parser_arguments){
    .nsweeps = 500000,
    .ngrids = 1248,
    .mag_output_int = 100,
    .grid_output_int = 100,
    .threads_per_block = 32,
    .gpu_device = 0,
    .gpu_method = 2,
    .beta = 0.54,
    .h = 0.07,
    .itask = 0,
    .output_file = (char*)"gridstates.bin",
    .input_file = NULL
  };

  parser_arguments args = parse_cl_arguments(argc, argv);

/*=================================
   Constants and variables
  =================================*/ 
  
  int L       = 64;            // Size of 2D Ising grid. LxL grid squares.
  int ngrids  = args.ngrids;             // Number of replicas of 2D grid to simulate
  int tot_nsweeps = args.nsweeps;       // Total number of MC sweeps to simulate on each grid

  int itask = args.itask;               // 0 = count samples which nucleate, 1 = compute committor

  int mag_output_int  = args.mag_output_int;   // Number of MC sweeps between calculation of magnetisation
  int grid_output_int = args.grid_output_int;  // Number of MC sweeps between dumps of grid to file

  double beta = args.beta;       // Inverse temperature
  double h = args.h;          // External field

  double dn_threshold = -0.90;         // Magnetisation at which we consider the system to have reached spin up state
  double up_threshold =  0.90;         // Magnetisation at which we consider the system to have reached spin down state

  //unsigned long rngseed = 2894203475;  // RNG seed (fixed for development/testing)
  unsigned long rngseed = (long)time(NULL);

  int threadsPerBlock = args.threads_per_block;            // Number of threads/replicas to run in each threadBlock
  int blocksPerGrid   = 1;             // Total number of threadBlocks
  int gpu_device = args.gpu_device;                 // GPU device to use
  int gpu_method = args.gpu_method;                  // Which MC sweep kernel to use

  char *output_filename = args.output_file;
  char *input_filename = args.input_file;

/*=================================
   Delete old output 
  ================================*/
  if (output_filename == NULL) {
    fprintf(stderr, "No output file set.\nExiting.\n");
    exit(EXIT_FAILURE);
  }
  remove(output_filename);


/*=================================
   Write output header 
  ================================*/
  if (itask==0) {
    printf("# isweep    nucleated fraction\n");
  }

/*=================================
   Initialise simulations
  =================================*/ 
  // Host copy of Ising grid configurations
  int *ising_grids = (int *)malloc(L*L*ngrids*sizeof(int));
  if (ising_grids==NULL){
    fprintf(stderr,"Error allocating memory for Ising grids!\n");
    exit(EXIT_FAILURE);
  }
  

  int i;
  int *grid_fate;  // stores pending(-1), reached B first (1) or reached A first (0)
  double pB;

  if (itask==0) {  // counting nucleated samples over time

    if (input_filename == NULL) {
      // Initialise as spin down  
      for (i=0;i<L*L*ngrids;i++) { ising_grids[i] = -1; }
    } else {
      // read input from file
      read_input_grid(L, ngrids, ising_grids, input_filename);
    }

  } else if (itask==1) {

    // check input filename is set
    if (input_filename == NULL) {
      // set to default if not already set
      input_filename = (char*)"gridinput.bin";
      printf("No input file specified, defaulting to %s", input_filename);
    }

    // Read from file
    read_input_grid(L, ngrids, ising_grids, input_filename);

    grid_fate = (int *)malloc(ngrids*sizeof(int));
    if (grid_fate==NULL) {
      printf("Error allocating memory for grid fates\n");
      exit(EXIT_FAILURE);
    }
    for (i=0;i<ngrids;i++) { grid_fate[i] = -1; } // all pending

  } else {

    printf("Error - unknown value of itask!");
    exit(EXIT_FAILURE);

  }


  // TODO - replace with configuration read from file

  // Initialise host RNG
  init_genrand(rngseed);

  // Precompute acceptance probabilities for flip moves
  preComputeProbs_cpu(beta, h);

  int *d_ising_grids;                    // Pointer to device grid configurations
  hiprandState *d_state;                  // Pointer to device RNG states
  int *d_neighbour_list;                 // Pointer to device neighbour lists

  // How many sweeps to run in each call
  int sweeps_per_call;
  sweeps_per_call = mag_output_int < grid_output_int ? mag_output_int : grid_output_int;

  if (run_gpu==true) {
    
    gpuInit(gpu_device); // Initialise GPU device(s)

    // Allocate threads to thread blocks
    blocksPerGrid = ngrids/threadsPerBlock;
    if (ngrids%threadsPerBlock!=0) { blocksPerGrid += 1; }

    // Device copy of Ising grid configurations
    gpuErrchk( hipMalloc(&d_ising_grids,L*L*ngrids*sizeof(int)) );

    // Populate from host copy
    gpuErrchk( hipMemcpy(d_ising_grids,ising_grids,L*L*ngrids*sizeof(int),hipMemcpyHostToDevice) );

    // Initialise GPU RNG
    gpuErrchk (hipMalloc((void **)&d_state, ngrids*sizeof(hiprandState)) );
    unsigned long long gpuseed = (unsigned long long)rngseed;
    init_gpurand<<<blocksPerGrid,threadsPerBlock>>>(gpuseed, ngrids, d_state);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    fprintf(stderr, "threadsPerBlock = %d, blocksPerGrid = %d\n",threadsPerBlock, blocksPerGrid);

    // Precompute acceptance probabilities for flip moves
    preComputeProbs_gpu(beta, h);

    // Neighbours
    gpuErrchk (hipMalloc((void **)&d_neighbour_list, L*L*4*sizeof(int)) );
    preComputeNeighbours_gpu(L, d_ising_grids, d_neighbour_list);

    // Test CUDA RNG (DEBUG)
    /*
    float   *testrnd = (float *)malloc(ngrids*sizeof(float));
    float *d_testrnd;
    gpuErrchk( hipMalloc(&d_testrnd, ngrids*sizeof(float)) );

    int trial;
    for (trial=0;trial<10;trial++){

      populate_random<<<blocksPerGrid,threadsPerBlock>>>(ngrids, d_testrnd, d_state);
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );
      gpuErrchk( hipMemcpy(testrnd, d_testrnd, ngrids*sizeof(float), hipMemcpyDeviceToHost) );

      for (i=0;i<ngrids;i++){
        printf("Random number on grid %d : %12.4f\n",i,testrnd[i]);
      }
  
  }

    free(testrnd);
    hipFree(d_testrnd);
    exit(EXIT_SUCCESS);
    */

  }

/*=================================
    Run simulations - CPU version
  =================================*/ 

  clock_t t1,t2;  // For measuring time taken
  int isweep;     // MC sweep loop counter
  int igrid;      // counter for loop over replicas



  if (run_cpu==true) {

    // Magnetisation of each grid
    double *magnetisation = (double *)malloc(ngrids*sizeof(double));
    if (magnetisation==NULL){
      fprintf(stderr,"Error allocating magnetisation array!\n");
      exit(EXIT_FAILURE);
    }

    t1 = clock();  // Start timer

    isweep = 0;
    while (isweep < tot_nsweeps){

      // Output grids to file
      if (isweep%grid_output_int==0){
        write_ising_grids(L, ngrids, ising_grids, isweep, output_filename);  
      }

      // Report magnetisations
      if (isweep%mag_output_int==0){
        for (igrid=0;igrid<ngrids;igrid++){
          compute_magnetisation_cpu(L, ising_grids, igrid, magnetisation);
          //printf("Magnetisation of grid %d at sweep %d = %8.4f\n",igrid, isweep, magnetisation[igrid]);
        }
        if ( itask == 0 ) { // Report how many samples have nucleated.
          int nnuc = 0;
          for (igrid=0;igrid<ngrids;igrid++){
            if ( magnetisation[igrid] > up_threshold ) nnuc++;
          }
          printf("%10d  %12.6f\n",isweep, (double)nnuc/(double)ngrids);
          if (nnuc==ngrids) break; // Stop if everyone has nucleated
        } else if ( itask == 1 ){

          // Statistics on fate of trajectories
          int nA=0, nB=0;
          for (igrid=0;igrid<ngrids;igrid++){
            //printf("grid_fate[%d] = %d\n",igrid, grid_fate[igrid]);
            if (grid_fate[igrid]==0 ) {
              nA++;
            } else if (grid_fate[igrid]==1 ) {
              nB++;
            } else {
              if ( magnetisation[igrid] > up_threshold ){
                grid_fate[igrid] = 1;
                nB++;
              } else if (magnetisation[igrid] < dn_threshold ){
                grid_fate[igrid] = 0;
                nA++;
              }
            } // fate
          } //grids

          // Monitor progress
          pB = (double)nB/(double)(nA+nB);
          printf("\r Sweep : %10d, Reached m = %6.2f : %4d , Reached m = %6.2f : %4d , Unresolved : %4d, pB = %10.6f",
           isweep, dn_threshold, nA, up_threshold, nB, ngrids-nA-nB,pB);
          fflush(stdout);
          if (nA + nB == ngrids) break; // all fates resolved
        } // task
      } 

      // MC Sweep - CPU
      for (igrid=0;igrid<ngrids;igrid++) {
        mc_sweep_cpu(L, ising_grids, igrid, beta, h, sweeps_per_call);
      }
      isweep += sweeps_per_call;

    }

    t2 = clock();  // Stop Timer

    printf("\n# Time taken on CPU = %f seconds\n",(double)(t2-t1)/(double)CLOCKS_PER_SEC);
    if (itask==1) { printf("pB estimate : %10.6f\n",pB); }

    // Release memory
    free(magnetisation);

  }

  /*=================================
    Run simulations - GPU version
  =================================*/ 
  if (run_gpu==true){

    // Host copy of magnetisation
    float *magnetisation = (float *)malloc(ngrids*sizeof(float));
    if (magnetisation==NULL){
      fprintf(stderr,"Error allocating magnetisation host array!\n");
      exit(EXIT_FAILURE);
    }

    // Device copy of magnetisation
    float *d_magnetisation;
    gpuErrchk( hipMalloc(&d_magnetisation,ngrids*sizeof(float)) );

    // Streams
    hipStream_t stream1;
    gpuErrchk( hipStreamCreate(&stream1) );

    hipStream_t stream2;
    gpuErrchk( hipStreamCreate(&stream2) );


    t1 = clock();  // Start Timer

    isweep = 0;
    while(isweep < tot_nsweeps){



      // Output grids to file
      if (isweep%grid_output_int==0){
        // Asynchronous - can happen while magnetisation is being computed in stream 2
        gpuErrchk( hipMemcpyAsync(ising_grids,d_ising_grids,L*L*ngrids*sizeof(int),hipMemcpyDeviceToHost,stream1) );
      }

      // Can compute manetisation while grids are copying
      if (isweep%mag_output_int==0){
        compute_magnetisation_gpu<<<blocksPerGrid, threadsPerBlock, 0, stream2>>>(L, ngrids, d_ising_grids, d_magnetisation);    
        gpuErrchk( hipMemcpyAsync(magnetisation,d_magnetisation,ngrids*sizeof(float),hipMemcpyDeviceToHost, stream2) );
      } 

      // MC Sweep - GPU
      gpuErrchk( hipStreamSynchronize(stream1) ); // Make sure copy completed before making changes

      if (gpu_method==0){
        mc_sweep_gpu<<<blocksPerGrid,threadsPerBlock,0,stream1>>>(L,d_state,ngrids,d_ising_grids,d_neighbour_list, (float)beta,(float)h,sweeps_per_call);
      } else if (gpu_method==1){
          size_t shmem_size = L*L*threadsPerBlock*sizeof(uint8_t)/8; // number of bytes needed to store grid as bits
          mc_sweep_gpu_bitrep<<<blocksPerGrid,threadsPerBlock,shmem_size,stream1>>>(L,d_state,ngrids,d_ising_grids, d_neighbour_list, (float)beta,(float)h,sweeps_per_call);
      } else if (gpu_method==2){
          size_t shmem_size = L*L*threadsPerBlock*sizeof(uint8_t)/8; // number of bytes needed to store grid as bits
          if (threadsPerBlock==32){
            mc_sweep_gpu_bitmap32<<<blocksPerGrid,threadsPerBlock,shmem_size,stream1>>>(L,d_state,ngrids,d_ising_grids, d_neighbour_list, (float)beta,(float)h,sweeps_per_call);
          } else if (threadsPerBlock==64){
            mc_sweep_gpu_bitmap64<<<blocksPerGrid,threadsPerBlock,shmem_size,stream1>>>(L,d_state,ngrids,d_ising_grids, d_neighbour_list, (float)beta,(float)h,sweeps_per_call);
          } else {
            printf("Invalid threadsPerBlock for gpu_method=2\n");
            exit(EXIT_FAILURE);
          } 
      } else {
        printf("Unknown gpu_method in ising.cu\n");
        exit(EXIT_FAILURE);
      }
      
      // Writing of the grids can be happening on the host while the device runs the mc_sweep kernel
      if (isweep%grid_output_int==0){
        write_ising_grids(L, ngrids, ising_grids, isweep, output_filename);  
      }

      // Write and report magnetisation - can also be happening while the device runs the mc_sweep kernel
      if (isweep%mag_output_int==0){
        gpuErrchk( hipStreamSynchronize(stream2) );  // Wait for copy to complete
        //for (igrid=0;igrid<ngrids;igrid++){
        //  printf("    %4d     %10d      %8.6f\n",igrid, isweep, magnetisation[igrid]);
        //}
        if ( itask == 0 ) { // Report how many samples have nucleated.
          int nnuc = 0;
          for (igrid=0;igrid<ngrids;igrid++){
            if ( magnetisation[igrid] > up_threshold ) nnuc++;
          }
          printf("%10d  %12.6f\n",isweep, (double)nnuc/(double)ngrids);
          if (nnuc==ngrids) break; // Stop if everyone has nucleated
        } else if ( itask == 1 ){

            // Statistics on fate of trajectories
            int nA=0, nB=0;
            for (igrid=0;igrid<ngrids;igrid++){
              if (grid_fate[igrid]==0 ) {
                nA++;
              } else if (grid_fate[igrid]==1 ) {
                nB++;
              } else {
                if ( magnetisation[igrid] > up_threshold ){
                  grid_fate[igrid] = 1;
                  nB++;
                } else if (magnetisation[igrid] < dn_threshold ){
                  grid_fate[igrid] = 0;
                  nA++;
                }
              } // fate
            } //grids

            // Monitor progress
            pB = (double)nB/(double)(nA+nB);
            printf("\r Sweep : %10d, Reached m = %6.2f : %4d , Reached m = %6.2f : %4d , Unresolved : %4d, pB = %10.6f",
            isweep, dn_threshold, nA, up_threshold, nB, ngrids-nA-nB,pB);
            fflush(stdout);
            if (nA + nB == ngrids) break; // all fates resolved
        
        } // task 
      }

      // Increment isweep
      isweep += sweeps_per_call;

      // Make sure all kernels updating the grids are finished before starting magnetisation calc
      gpuErrchk( hipStreamSynchronize(stream1) );
      gpuErrchk( hipPeekAtLastError() );

    }

    // Ensure all threads finished before stopping timer
    gpuErrchk( hipDeviceSynchronize() )

    t2 = clock();

    printf("\n# Time taken on GPU = %f seconds\n",(double)(t2-t1)/(double)CLOCKS_PER_SEC);
    if (itask==1) { printf("pB estimate : %10.6f\n",pB); }

    // Destroy streams
    gpuErrchk( hipStreamDestroy(stream1) );
    gpuErrchk( hipStreamDestroy(stream2) );


    // Free magnetisation arrays
    free(magnetisation);
    gpuErrchk( hipFree(d_magnetisation) );

  }


/*=================================================
    Tidy up memory used in both GPU and CPU paths
  =================================================*/ 
  free(ising_grids);
  if (run_gpu==true) {
    gpuErrchk( hipFree(d_ising_grids) );
    gpuErrchk( hipFree(d_state) );
    gpuErrchk( hipFree(d_neighbour_list) );
  }

  return EXIT_SUCCESS;

}
