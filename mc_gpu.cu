#include "hip/hip_runtime.h"
// -*- mode: C -*-

#include "mc_gpu.h"
#include <stdio.h>

// Cache of acceptance probabilities 
__constant__ float d_Pacc[20];   // gpu constant memory

// Dynamic shared memory for storage of bits
extern __shared__ uint8_t shared_grid[];

// populate acceptance probabilities
void preComputeProbs_gpu(double beta, double h) {

    float *h_Pacc=(float *)malloc(20*sizeof(float));

    int s, nsum, index;  
    for (s=-1;s<2;s=s+2){
      for (nsum=-4;nsum<5;nsum=nsum+2){
        index = 5*(s+1) + nsum + 4;
        h_Pacc[index] = 2.0f*(float)s*((float)nsum+(float)h);
        h_Pacc[index] = expf(-(float)beta*h_Pacc[index]); // single precision version for GPU
      }
    }
  
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_Pacc), h_Pacc, 20*sizeof(float),0, hipMemcpyHostToDevice ) );
    free(h_Pacc);

  }  

void preComputeNeighbours_gpu(const int L, int *d_ising_grids, int *d_neighbour_list){

// These could probably be cached in shared memory since they are the same for all threads.

int *h_neighbour_list = (int *)malloc(L*L*4*sizeof(int));

int spin_index;
for (spin_index=0;spin_index<L*L;spin_index++){

  int row = spin_index/L;
  int col = spin_index%L;

  h_neighbour_list[4*(row*L+col) + 0] = L*((row+1)%L) + col;
  h_neighbour_list[4*(row*L+col) + 1] = L*((row+L-1)%L) + col;
  h_neighbour_list[4*(row*L+col) + 2] = L*row + (col+1)%L;
  h_neighbour_list[4*(row*L+col) + 3] = L*row + (col+L-1)%L;

}

gpuErrchk( hipMemcpy(d_neighbour_list, h_neighbour_list, 4*L*L*sizeof(int),hipMemcpyHostToDevice ) );

free(h_neighbour_list); 

/// Also store a version in constant memory
uint8_t *hc_next = (uint8_t *)malloc(MAXL*sizeof(uint8_t));
uint8_t *hc_prev = (uint8_t *)malloc(MAXL*sizeof(uint8_t));

for (spin_index=0;spin_index<L;spin_index++){

  hc_next[spin_index] = (spin_index+1)%L;
  hc_prev[spin_index] = (spin_index+L-1)%L;

}

gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dc_next), hc_next, MAXL*sizeof(uint8_t),0, hipMemcpyHostToDevice ) );
gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dc_prev), hc_prev, MAXL*sizeof(uint8_t),0, hipMemcpyHostToDevice ) );
  
free(hc_next); 
free(hc_prev);

}


// sweep on the gpu - default version
__global__ void mc_sweep_gpu(const int L, hiprandState *state, const int ngrids, int *d_ising_grids, int *d_neighbour_list, const float beta, const float h, int nsweeps) {

  int idx = threadIdx.x+blockIdx.x*blockDim.x;
  int index;

  if (idx < ngrids) {

    // local copy of RNG state for current threads 
    hiprandState localState = state[idx];

    int N = L*L;
    // Avoid rounding errors after creating random numbers by ensuring out max
    // is upto 1.0f not up to 1.0f + FLT_EPSILON
    float shrink = (1.0f - FLT_EPSILON)*(float)N;

    // Pointer to local grid
    int *loc_grid = &d_ising_grids[idx*N]; // pointer to device global memory 


    int imove, my_idx, spin, n1, n2, n3, n4, row, col;
    for (imove=0;imove<N*nsweeps;imove++){

      my_idx = __float2int_rd(shrink*hiprand_uniform(&localState));

      row = my_idx/L;
      col = my_idx%L;

      spin = loc_grid[my_idx];

      // find neighbours, periodic boundary conditions. D,U,L,R
      n1 = loc_grid[L*((row+1)%L) + col];
      n2 = loc_grid[L*((row+L-1)%L) + col];
      n3 = loc_grid[L*row + (col+1)%L];
      n4 = loc_grid[L*row + (col+L-1)%L];

      //n_sum = 4;
      index = 5*(spin+1) + n1+n2+n3+n4 + 4;

      // The store back to global memory, not the branch or the RNG generation
      // seems to be the killer here.
      if (hiprand_uniform(&localState) < d_Pacc[index] ) {
          // accept
          loc_grid[my_idx] = -1*spin;
      } 

      // TODO: 
      // Attempt a shared memory cache of changes then run the calculation 
      // until this cache would do a register overflow and then do a write 
      // global memory. This would need a list in shared memory that is 
      // ngrids*bytes_to_store(up to ngrids*N_max)*number_of_cached_changes_per_grid
      // This assumes any change would be a flip thus when we calculate the spins
      // we simply do a flip back if hitting a change.
      // additionally for cacheing we could use two threads per block and use 
      // one to do the copying and one to do the calculation async. The chance of a 
      // collision and a lock is low it will happen but not enough to be a bottleneck?
      
      // Try avoiding the branch entirely - this seems quite slow
      //diff = hiprand_uniform(&localState) - d_Pacc[index] ;
      //spin = spin * lrintf(copysignf(1.0f,diff)); 
      //loc_grid[my_idx] = spin;

      // This is even slower (and has a hidden branch)
      //diff = hiprand_uniform(&localState) - d_Pacc[index] ;
      //spin = signbit(diff) ? -1*spin : spin ;
      //loc_grid[my_idx] = spin;
      
    } //end for


    // Copy local data back to device global memory
    state[idx] = localState;

  }

  return;

}

inline __device__ int bits_to_ints(uint8_t* grid, int index){
  // L must be the number of entries in each row here. Important if ever adapting to non-square grids.

  uint8_t one = 1U;
  int blookup[2] = {-1, 1};

  //nt ibyte = (index)/8;
  //int ibit  = (index)%8;

  // From CUDA-C best practices guide
  int ibyte = index >> 3;
  int ibit  = index & 7 ;

  return blookup[(grid[ibyte] >> ibit) & one]; 

}

// sweep on the gpu - packs a cache of the current grid into on-GPU shared memory
// for efficiency, and using a single bit represenation to acheive this. Mustn't
// be used if L*L*threadsPerBlock/8 > SHGRIDSIZE. 
__global__ void mc_sweep_gpu_bitrep(const int L, hiprandState *state, const int ngrids, int *d_ising_grids, int *d_neighbour_list, const float beta, const float h, int nsweeps) {

  const int blookup[2] = {-1, 1};

  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  if (idx < ngrids) {

    // local copy of RNG state for current threads 
    hiprandState localState = state[idx];

    // how many bytes do we need per thread to store L*L spins as single bytes
    int nbytes = L*L/8;
    if ( (L*L)%8 !=0 ) { nbytes ++; }

    // If nbytes x threadsPerBlock is greater than the shared memory size
    // then we've got a problem, but should have had a kernel launch 
    // failure before getting this far so not checking that here.

    // Pointer to part of this storage used by the current thread
    uint8_t *loc_grid = &shared_grid[nbytes*threadIdx.x];
    uint8_t one  = 1U;
    uint8_t zero = 0U;

    // zero the local grid
    int ibyte;
    for (ibyte=0;ibyte<nbytes;ibyte++){ loc_grid[ibyte] = zero ; }

    // Fill this with the current state of the grid to be manipulated by this thread
    int ibit=0, spin;
    ibyte = 0;
    for (spin=0;spin<L*L;spin++){ 
        if ( d_ising_grids[L*L*idx + spin] == 1 ) {
          loc_grid[ibyte] |= one << ibit ;
        }
        ibit++;
        if (ibit==8) {
            ibit=0;
            ibyte++;
        }
    }
     

    float shrink = 1.0f - FLT_EPSILON;
    int imove, row, col, index,  n1, n2, n3, n4;
    uint16_t spin_index;

    for (imove=0;imove<L*L*nsweeps;imove++){

      spin_index = __float2uint_rd((float)L*L*shrink*hiprand_uniform(&localState));
      row = spin_index/L;
      col = spin_index%L;
      
      //spin = bits_to_ints(loc_grid, spin_index);
      spin = blookup[(loc_grid[spin_index >> 3] >> (spin_index & 7)) & one];

      // find neighbours
      n1 = L*((row+1)%L) + col;
      n1 = blookup[(loc_grid[n1 >> 3] >> (n1 & 7)) & one];

      n2 = L*((row+L-1)%L) + col;
      n2 = blookup[(loc_grid[n2 >> 3] >> (n2 & 7)) & one];

      n3 = L*row + (col+1)%L;
      n3 = blookup[(loc_grid[n3 >> 3] >> (n3 & 7)) & one];

      n4 = L*row + (col+L-1)%L;
      n4 = blookup[(loc_grid[n4 >> 3] >> (n4 & 7)) & one];

      //n_sum = 4;
      index = 5*(spin+1) + n1 + n2 + n3 + n4 + 4;

      if (hiprand_uniform(&localState) < d_Pacc[index] ) {
          // accept - toggle bit
          ibyte = spin_index >> 3;
          index = spin_index & 7;

          //ibyte = spin_index / 8;
          //index = spin_index % 8; 
          loc_grid[ibyte] ^= one << index;
      }
      
      
    } //end for


    // Copy local data back to device global memory
    state[idx] = localState;

    //for (row=0;row<L;row++){
    //  for (col=0;col<L;col++){
    for (spin_index=0;spin_index<L*L;spin_index++){
      d_ising_grids[L*L*idx + spin_index] = blookup[(loc_grid[spin_index >> 3] >> (spin_index & 7)) & one];
      //}
    }

    

  }

  return;

}

// Similar to mc_sweep_gpu_bitrep, but maps each thread in a block of 32 threads to a 
// fixed bit in a datatype of size 4 bytes for faster addressing.
__global__ void mc_sweep_gpu_bitmap32(const int L, hiprandState *state, const int ngrids, int *d_ising_grids, int *d_neighbour_list, const float beta, const float h, int nsweeps) {

  const int llookup[2] = {-1, 1};

  // Shared memory for storage of bits
  uint32_t *bit_grid = (uint32_t *)&shared_grid[0];
  uint32_t one  = 1U;
  uint32_t zero = 0U;

  // Location in global memory where grids for the current block are stored
  int *block_grid = &d_ising_grids[L*L*blockIdx.x*blockDim.x];

  // Populate from global memory, ensuring that uint32_t is only written to by a single thread.
  int ispin,spin,ibit;
  for (ispin=threadIdx.x;ispin<L*L;ispin+=blockDim.x){
    bit_grid[ispin] = zero; 
    for (ibit=0;ibit<blockDim.x;ibit++){
      spin = block_grid[ibit*L*L + ispin];
      if ( spin == 1 ) {
        bit_grid[ispin] ^= one << ibit;
      }
    }
  }

  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  if (idx < ngrids) {

    // local copy of RNG state for current threads 
    hiprandState localState = state[idx];
 
    int N=L*L;
    float shrink = (1.0f - FLT_EPSILON)*(float)N;
    //float shrink = (1.0f - FLT_EPSILON);
    int imove, row, col, index, my_idx, n1, n2 , n3, n4;

    for (imove=0;imove<N*nsweeps;imove++){

      my_idx = __float2int_rd(shrink*hiprand_uniform(&localState));
      row = my_idx/L;
      col = my_idx%L;
 
      spin = llookup[(bit_grid[my_idx] >> threadIdx.x) & one];
      
      // find neighbours
      n1 = llookup[(bit_grid[L*((row+1)%L) + col] >> threadIdx.x) & one];
      n2 = llookup[(bit_grid[L*((row+L-1)%L) + col] >> threadIdx.x) & one];
      n3 = llookup[(bit_grid[L*row + (col+1)%L] >> threadIdx.x) & one];
      n4 = llookup[(bit_grid[L*row + (col+L-1)%L] >> threadIdx.x) & one];

      //n_sum = 4;
      index = 5*(spin+1) + n1 + n2 + n3 + n4 + 4;

      if (hiprand_uniform(&localState) < d_Pacc[index] ) {
          // accept - toggle bit
          //bit_grid[my_idx] ^= one << threadIdx.x;
          atomicXor(&bit_grid[my_idx],one << threadIdx.x);

      } 
      
      
    } //end for

    // Copy local data back to device global memory
    state[idx] = localState;

    for (row=0;row<L;row++){
      for (col=0;col<L;col++){
        my_idx = L*row + col;
        d_ising_grids[N*idx+my_idx] = llookup[(bit_grid[my_idx] >> threadIdx.x) & one];
      }
    }

  }

  return;

}

__global__ void mc_sweep_gpu_bitmap64(const int L, hiprandState *state, const int ngrids, int *d_ising_grids, int *d_neighbour_list, const float beta, const float h, int nsweeps) {

  const int llookup[2] = {-1, 1};

  // Shared memory for storage of bits
  uint64_t *bit_grid = (uint64_t *)&shared_grid[0];
  uint64_t one  = 1U;
  uint64_t zero = 0U;

  // Location in global memory where grids for the current block are stored
  int *block_grid = &d_ising_grids[L*L*blockIdx.x*blockDim.x];

  // Populate from global memory, ensuring that uint64_t is only written to by a single thread.
  int ispin,spin,ibit;
  for (ispin=threadIdx.x;ispin<L*L;ispin+=blockDim.x){
    bit_grid[ispin] = zero; 
    for (ibit=0;ibit<blockDim.x;ibit++){
      spin = block_grid[ibit*L*L + ispin];
      if ( spin == 1 ) {
        bit_grid[ispin] ^= one << ibit;
      }
    }
  }

  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  if (idx < ngrids) {

    // local copy of RNG state for current threads 
    hiprandState localState = state[idx];
  
    int N=L*L;
    float shrink = (1.0f - FLT_EPSILON)*(float)N;
    //float shrink = (1.0f - FLT_EPSILON);
    int imove, row, col, index, my_idx, n1, n2 , n3, n4;

    for (imove=0;imove<N*nsweeps;imove++){

      my_idx = __float2int_rd(shrink*hiprand_uniform(&localState));
      row = my_idx/L;
      col = my_idx%L;
 
      spin = llookup[(bit_grid[my_idx] >> threadIdx.x) & one];
      
      // find neighbours
      n1 = llookup[(bit_grid[L*((row+1)%L) + col] >> threadIdx.x) & one];
      n2 = llookup[(bit_grid[L*((row+L-1)%L) + col] >> threadIdx.x) & one];
      n3 = llookup[(bit_grid[L*row + (col+1)%L] >> threadIdx.x) & one];
      n4 = llookup[(bit_grid[L*row + (col+L-1)%L] >> threadIdx.x) & one];

      //n_sum = 4;
      index = 5*(spin+1) + n1 + n2 + n3 + n4 + 4;

      if (hiprand_uniform(&localState) < d_Pacc[index] ) {
          // accept - toggle bit
          bit_grid[my_idx] ^= one << threadIdx.x;
          //atomicXor(&bit_grid[my_idx],one << threadIdx.x);  // unsupported for uint64_t
      } 
      
      
    } //end for

    // Copy local data back to device global memory
    state[idx] = localState;

    for (row=0;row<L;row++){
      for (col=0;col<L;col++){
        my_idx = L*row + col;
        d_ising_grids[N*idx+my_idx] = llookup[(bit_grid[my_idx] >> threadIdx.x) & one];
      }
    }

  }

  return;

}

// compute magnetisation on the gpu
__global__ void compute_magnetisation_gpu(const int L, const int ngrids, int *d_ising_grids, float *d_magnetisation) {

  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  if (idx < ngrids) {

    int *loc_grid = &d_ising_grids[idx*L*L]; // pointer to device global memory

    float m = 0.0f;

    int i;
    for (i=0;i<L*L;i++) { m += loc_grid[i]; }
    d_magnetisation[idx] = m/(float)(L*L);

  }

  return;

}