#include "hip/hip_runtime.h"
// -*- mode: C -*-

#include "mc_gpu.h"
#include <stdio.h>
#include <cstdint>
#include <iostream>


// Cache of acceptance probabilities 
__constant__ float d_Pacc[20];   // gpu constant memory

// Dynamic shared memory for storage of bits
extern __shared__ uint8_t shared_grid[];

// populate acceptance probabilities
void preComputeProbs_gpu(double beta, double h) {

    float *h_Pacc=(float *)malloc(20*sizeof(float));

    int s, nsum, index;  
    for (s=-1;s<2;s=s+2){
      for (nsum=-4;nsum<5;nsum=nsum+2){
        index = 5*(s+1) + nsum + 4;
        h_Pacc[index] = 2.0f*(float)s*((float)nsum+(float)h);
        h_Pacc[index] = expf(-(float)beta*h_Pacc[index]); // single precision version for GPU
      }
    }
  
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_Pacc), h_Pacc, 20*sizeof(float),0, hipMemcpyHostToDevice ) );
    free(h_Pacc);

  }  

void preComputeNeighbours_gpu(const int L, int *d_ising_grids, int *d_neighbour_list){

// These could probably be cached in shared memory since they are the same for all threads.

int *h_neighbour_list = (int *)malloc(L*L*4*sizeof(int));

int spin_index;
for (spin_index=0;spin_index<L*L;spin_index++){

  int row = spin_index/L;
  int col = spin_index%L;

  h_neighbour_list[4*(row*L+col) + 0] = L*((row+1)%L) + col;
  h_neighbour_list[4*(row*L+col) + 1] = L*((row+L-1)%L) + col;
  h_neighbour_list[4*(row*L+col) + 2] = L*row + (col+1)%L;
  h_neighbour_list[4*(row*L+col) + 3] = L*row + (col+L-1)%L;

}

gpuErrchk( hipMemcpy(d_neighbour_list, h_neighbour_list, 4*L*L*sizeof(int),hipMemcpyHostToDevice ) );

free(h_neighbour_list); 

/// Also store a version in constant memory
uint8_t *hc_next = (uint8_t *)malloc(MAXL*sizeof(uint8_t));
uint8_t *hc_prev = (uint8_t *)malloc(MAXL*sizeof(uint8_t));

for (spin_index=0;spin_index<L;spin_index++){

  hc_next[spin_index] = (spin_index+1)%L;
  hc_prev[spin_index] = (spin_index+L-1)%L;

}
}

__host__ __device__ bp_cell_id get_bp_cell(const int L, int16_t row, int16_t col){
  bp_cell_id c;
  int64_t bit = row * L + col;
  c.byte = bit/8;
  c.bit = 1 << bit%8;
  return c;
}

void preComputeNeighbours_gpu_bp(const int L, int *d_ising_grids, bp_cell_id *d_neighbour_list){

// These could probably be cached in shared memory since they are the same for all threads.

  bp_cell_id *h_neighbour_list = new bp_cell_id[L*L*4];

  int spin_index;
  for (spin_index=0;spin_index<L*L;spin_index++){

    int row = spin_index/L;
    int col = spin_index%L;
    //Get offset cells
    int rowp = (row+1)%L, rowm = (row+L-1)%L;
    int colp = (col+1)%L, colm = (row+L-1)%L;

    h_neighbour_list[4*(row*L+col) + 0] = get_bp_cell(L,rowp, col);
    h_neighbour_list[4*(row*L+col) + 1] = get_bp_cell(L,rowm, col);
    h_neighbour_list[4*(row*L+col) + 2] = get_bp_cell(L,row, colp);
    h_neighbour_list[4*(row*L+col) + 3] = get_bp_cell(L,row, colm);

  }

  gpuErrchk( hipMemcpy(d_neighbour_list, h_neighbour_list, 4*L*L*sizeof(bp_cell_id),hipMemcpyHostToDevice ) );

  delete[] h_neighbour_list;

/*  /// Also store a version in constant memory
  uint8_t *hc_next = new uint8_t[MAXL];
  uint8_t *hc_prev = new uint8_t[MAXL];

  for (spin_index=0;spin_index<L;spin_index++){

    hc_next[spin_index] = (spin_index+1)%L;
    hc_prev[spin_index] = (spin_index+L-1)%L;

  }

  gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dc_next), hc_next, MAXL*sizeof(uint8_t),0, hipMemcpyHostToDevice ) );
  gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dc_prev), hc_prev, MAXL*sizeof(uint8_t),0, hipMemcpyHostToDevice ) );
  
  delete[] hc_next; 
  delete[] hc_prev;*/

  }


// sweep on the gpu - default version
__global__ void mc_sweep_gpu(const int L, hiprandState *state, const int ngrids, int *d_ising_grids, int *d_neighbour_list, const float beta, const float h, int nsweeps) {

  int idx = threadIdx.x+blockIdx.x*blockDim.x;
  int index;

  if (idx < ngrids) {

    // local copy of RNG state for current threads 
    hiprandState localState = state[idx];

    int N = L*L;
    float shrink = (1.0f - FLT_EPSILON)*(float)N;

    // Pointer to local grid
    int *loc_grid = &d_ising_grids[idx*N]; // pointer to device global memory 


    int imove, my_idx, spin, n1, n2, n3, n4, row, col;
    for (imove=0;imove<N*nsweeps;imove++){

      my_idx = __float2int_rd(shrink*hiprand_uniform(&localState));

      row = my_idx/L;
      col = my_idx%L;

      spin = loc_grid[my_idx];

      // find neighbours
      n1 = loc_grid[L*((row+1)%L) + col];
      n2 = loc_grid[L*((row+L-1)%L) + col];
      n3 = loc_grid[L*row + (col+1)%L];
      n4 = loc_grid[L*row + (col+L-1)%L];

      //n_sum = 4;
      index = 5*(spin+1) + n1+n2+n3+n4 + 4;

      // The store back to global memory, not the branch or the RNG generation
      // seems to be the killer here.
      if (hiprand_uniform(&localState) < d_Pacc[index] ) {
          // accept
          loc_grid[my_idx] = -1*spin;
      } 
      
      // Try avoiding the branch entirely - this seems quite slow
      //diff = hiprand_uniform(&localState) - d_Pacc[index] ;
      //spin = spin * lrintf(copysignf(1.0f,diff)); 
      //loc_grid[my_idx] = spin;

      // This is even slower (and has a hidden branch)
      //diff = hiprand_uniform(&localState) - d_Pacc[index] ;
      //spin = signbit(diff) ? -1*spin : spin ;
      //loc_grid[my_idx] = spin;
      
    } //end for


    // Copy local data back to device global memory
    state[idx] = localState;

  }

  return;

}

inline __device__ int bits_to_ints(uint8_t* grid, int index){
  // L must be the number of entries in each row here. Important if ever adapting to non-square grids.

  uint8_t one = 1U;
  int blookup[2] = {-1, 1};

  //nt ibyte = (index)/8;
  //int ibit  = (index)%8;

  // From CUDA-C best practices guide
  int ibyte = index >> 3;
  int ibit  = index & 7 ;

  return blookup[(grid[ibyte] >> ibit) & one]; 

}

// sweep on the gpu - packs a cache of the current grid into on-GPU shared memory
// for efficiency, and using a single bit represenation to acheive this. Mustn't
// be used if L*L*threadsPerBlock/8 > SHGRIDSIZE. 
__global__ void mc_sweep_gpu_bitrep(const int L, hiprandState *state, const int ngrids, int *d_ising_grids, int *d_neighbour_list, const float beta, const float h, int nsweeps) {

  const int blookup[2] = {-1, 1};

  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  if (idx < ngrids) {

    // local copy of RNG state for current threads 
    hiprandState localState = state[idx];

    // how many bytes do we need per thread to store L*L spins as single bytes
    int nbytes = L*L/8;
    if ( (L*L)%8 !=0 ) { nbytes ++; }

    // If nbytes x threadsPerBlock is greater than the shared memory size
    // then we've got a problem, but should have had a kernel launch 
    // failure before getting this far so not checking that here.

    // Pointer to part of this storage used by the current thread
    uint8_t *loc_grid = &shared_grid[nbytes*threadIdx.x];
    uint8_t one  = 1U;
    uint8_t zero = 0U;

    // zero the local grid
    int ibyte;
    for (ibyte=0;ibyte<nbytes;ibyte++){ loc_grid[ibyte] = zero ; }

    // Fill this with the current state of the grid to be manipulated by this thread
    int ibit=0, spin;
    ibyte = 0;
    for (spin=0;spin<L*L;spin++){ 
        if ( d_ising_grids[L*L*idx + spin] == 1 ) {
          loc_grid[ibyte] |= one << ibit ;
        }
        ibit++;
        if (ibit==8) {
            ibit=0;
            ibyte++;
        }
    }
     

    float shrink = 1.0f - FLT_EPSILON;
    int imove, row, col, index,  n1, n2, n3, n4;
    uint16_t spin_index;

    for (imove=0;imove<L*L*nsweeps;imove++){

      spin_index = __float2uint_rd((float)L*L*shrink*hiprand_uniform(&localState));
      row = spin_index/L;
      col = spin_index%L;
      
      //spin = bits_to_ints(loc_grid, spin_index);
      spin = blookup[(loc_grid[spin_index >> 3] >> (spin_index & 7)) & one];

      // find neighbours
      n1 = L*((row+1)%L) + col;
      n1 = blookup[(loc_grid[n1 >> 3] >> (n1 & 7)) & one];

      n2 = L*((row+L-1)%L) + col;
      n2 = blookup[(loc_grid[n2 >> 3] >> (n2 & 7)) & one];

      n3 = L*row + (col+1)%L;
      n3 = blookup[(loc_grid[n3 >> 3] >> (n3 & 7)) & one];

      n4 = L*row + (col+L-1)%L;
      n4 = blookup[(loc_grid[n4 >> 3] >> (n4 & 7)) & one];

      //n_sum = 4;
      index = 5*(spin+1) + n1 + n2 + n3 + n4 + 4;

      if (hiprand_uniform(&localState) < d_Pacc[index] ) {
          // accept - toggle bit
          ibyte = spin_index >> 3;
          index = spin_index & 7;

          //ibyte = spin_index / 8;
          //index = spin_index % 8; 
          loc_grid[ibyte] ^= one << index;
      } 
      
      
    } //end for


    // Copy local data back to device global memory
    state[idx] = localState;

    //for (row=0;row<L;row++){
    //  for (col=0;col<L;col++){
    for (spin_index=0;spin_index<L*L;spin_index++){
      d_ising_grids[L*L*idx + spin_index] = blookup[(loc_grid[spin_index >> 3] >> (spin_index & 7)) & one];
      //}
    }

    

  }

  return;

}

//Rotate packing so that data is packed with each threads data separate in memory
__global__ void mc_sweep_gpu_bitpacked(const int L, hiprandState *state, const int ngrids, int *d_ising_grids, bp_cell_id *d_neighbour_list, const float beta, const float h, int nsweeps) {

  const int llookup[2] = {-1, 1};

  // Shared memory for storage of bits
  uint8_t *bit_grid = &shared_grid[(1+(L*L-1)/8)*threadIdx.x];

  uint8_t one  = 1U;
  uint8_t zero = 0U;

  // Location in global memory where grids for the current block are stored
  int *block_grid = &d_ising_grids[L*L*(blockIdx.x*blockDim.x+threadIdx.x)];


  // Populate from global memory, ensuring that uint32_t is only written to by a single thread.
  int ispin,spin,ibit;
//  printf("%i : %i : %i\n", blockIdx.x, threadIdx.x, L*L*threadIdx.x);
  for (ispin=0;ispin<L*L;ispin+=8){
    bit_grid[ispin/8] = zero;
    for (ibit=0;ibit<8;ibit++){
      spin = block_grid[ispin + ibit];
      if ( spin == 1 ) {
        bit_grid[ispin/8] |= one;
      }
      bit_grid[ispin/8]=bit_grid[ispin/8] << 1;
    }
  }

  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  if (idx < ngrids) {
    // local copy of RNG state for current threads
    hiprandState localState = state[idx];

    int N=L*L;
    float shrink = (1.0f - FLT_EPSILON)*(float)N;
    //float shrink = (1.0f - FLT_EPSILON);
    int imove, row, col, index, my_idx, n1, n2 , n3, n4;

    auto idc = get_bp_cell(L,row,col);
    auto id=idc;

    for (imove=0;imove<N*nsweeps;imove++){

      my_idx = __float2int_rd(shrink*hiprand_uniform(&localState));
      row = my_idx/L;
      col = my_idx%L;
      int rowp = (row+1)%L, rowm = (row+L-1)%L;
      int colp = (col+1)%L, colm = (row+L-1)%L;
      auto idc = get_bp_cell(L,row,col);

      spin = llookup[(bit_grid[idc.byte] & idc.bit)>0];

      // find neighbours
//      auto id = d_neighbour_list[4*(row*L+col) + 0];
      auto id = get_bp_cell(L,rowp,col);
      n1 = llookup[(bit_grid[id.byte] & id.bit)>0];
      id = get_bp_cell(L,rowm,col);
      n2 = llookup[(bit_grid[id.byte] & id.bit)>0];
      id = get_bp_cell(L,row,colp);
      n3 = llookup[(bit_grid[id.byte] & id.bit)>0];
      id = get_bp_cell(L,row,colm);
      n4 = llookup[(bit_grid[id.byte] & id.bit)>0];

      //n_sum = 4;
      index = 5*(spin+1) + n1 + n2 + n3 + n4 + 4;

      if (hiprand_uniform(&localState) < d_Pacc[index] ) {
	      bit_grid[idc.byte] ^= idc.bit;
      }


    } //end for

    // Copy local data back to device global memory
    state[idx] = localState;

    int l_idx=-1;
    int ct=0;
    for (row=0;row<L;row++){
      for (col=0;col<L;col++){
	l_idx++;
	auto id = get_bp_cell(L,row,col);
	block_grid[l_idx] = llookup[(bit_grid[id.byte] & id.bit)>0];
	ct+=block_grid[l_idx];
      }
    }
  }

  return;

}



// Similar to mc_sweep_gpu_bitrep, but maps each thread in a block of 32 threads to a 
// fixed bit in a datatype of size 4 bytes for faster addressing.
__global__ void mc_sweep_gpu_bitmap32(const int L, hiprandState *state, const int ngrids, int *d_ising_grids, int *d_neighbour_list, const float beta, const float h, int nsweeps) {

  const int llookup[2] = {-1, 1};

  // Shared memory for storage of bits
  uint32_t *bit_grid = (uint32_t *)&shared_grid[0];
  uint32_t one  = 1U;
  uint32_t zero = 0U;

  // Location in global memory where grids for the current block are stored
  int *block_grid = &d_ising_grids[L*L*blockIdx.x*blockDim.x];

  // Populate from global memory, ensuring that uint32_t is only written to by a single thread.
  int ispin,spin,ibit;
  for (ispin=threadIdx.x;ispin<L*L;ispin+=blockDim.x){
    bit_grid[ispin] = zero; 
    for (ibit=0;ibit<blockDim.x;ibit++){
      spin = block_grid[ibit*L*L + ispin];
      if ( spin == 1 ) {
        bit_grid[ispin] |= one << ibit;
      }
    }
  }

  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  if (idx < ngrids) {

    // local copy of RNG state for current threads 
    hiprandState localState = state[idx];
 
    int N=L*L;
    float shrink = (1.0f - FLT_EPSILON)*(float)N;
    //float shrink = (1.0f - FLT_EPSILON);
    int imove, row, col, index, my_idx, n1, n2 , n3, n4;

    for (imove=0;imove<N*nsweeps;imove++){

      my_idx = __float2int_rd(shrink*hiprand_uniform(&localState));
      row = my_idx/L;
      col = my_idx%L;
 
      spin = llookup[(bit_grid[my_idx] >> threadIdx.x) & one];
      
      // find neighbours
      n1 = llookup[(bit_grid[L*((row+1)%L) + col] >> threadIdx.x) & one];
      n2 = llookup[(bit_grid[L*((row+L-1)%L) + col] >> threadIdx.x) & one];
      n3 = llookup[(bit_grid[L*row + (col+1)%L] >> threadIdx.x) & one];
      n4 = llookup[(bit_grid[L*row + (col+L-1)%L] >> threadIdx.x) & one];

      //n_sum = 4;
      index = 5*(spin+1) + n1 + n2 + n3 + n4 + 4;

/*      int val = hiprand_uniform(&localState) < d_Pacc[index];
      int obg = bit_grid[my_idx];
      atomicXor(&bit_grid[my_idx],one << threadIdx.x);
      bit_grid[my_idx] = bit_grid[my_idx] * val + obg * (1-val);*/

//      atomicCAS(&bit_grid[my_idx],hiprand_uniform(&localState) < d_Pacc[index], bit_grid[my_idx] ^ one << threadIdx.x);


      if (hiprand_uniform(&localState) < d_Pacc[index] ) {
//          if (hiprand_uniform(&localState) < df ) {
          // accept - toggle bit
          //bit_grid[my_idx] ^= one << threadIdx.x;
          atomicXor(&bit_grid[my_idx],one << threadIdx.x);
//	    atomicCAS(&bit_grid[my_idx],hiprand_uniform(&localState) < d_Pacc[index], bit_grid[my_idx] ^ one << threadIdx.x);

      }
      
      
    } //end for

    // Copy local data back to device global memory
    state[idx] = localState;

    for (row=0;row<L;row++){
      for (col=0;col<L;col++){
        my_idx = L*row + col;
        d_ising_grids[N*idx+my_idx] = llookup[(bit_grid[my_idx] >> threadIdx.x) & one];
      }
    }
  }

  return;

}

__global__ void mc_sweep_gpu_bitmap64(const int L, hiprandState *state, const int ngrids, int *d_ising_grids, int *d_neighbour_list, const float beta, const float h, int nsweeps) {

  const int llookup[2] = {-1, 1};

  // Shared memory for storage of bits
  uint64_t *bit_grid = (uint64_t *)&shared_grid[0];
  uint64_t one  = 1U;
  uint64_t zero = 0U;

  // Location in global memory where grids for the current block are stored
  int *block_grid = &d_ising_grids[L*L*blockIdx.x*blockDim.x];

  // Populate from global memory, ensuring that uint64_t is only written to by a single thread.
  int ispin,spin,ibit;
  for (ispin=threadIdx.x;ispin<L*L;ispin+=blockDim.x){
    bit_grid[ispin] = zero; 
    for (ibit=0;ibit<blockDim.x;ibit++){
      spin = block_grid[ibit*L*L + ispin];
      if ( spin == 1 ) {
        bit_grid[ispin] ^= one << ibit;
      }
    }
  }

  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  if (idx < ngrids) {

    // local copy of RNG state for current threads 
    hiprandState localState = state[idx];
  
    int N=L*L;
    float shrink = (1.0f - FLT_EPSILON)*(float)N;
    //float shrink = (1.0f - FLT_EPSILON);
    int imove, row, col, index, my_idx, n1, n2 , n3, n4;

    for (imove=0;imove<N*nsweeps;imove++){

      my_idx = __float2int_rd(shrink*hiprand_uniform(&localState));
      row = my_idx/L;
      col = my_idx%L;
 
      spin = llookup[(bit_grid[my_idx] >> threadIdx.x) & one];
      
      // find neighbours
      n1 = llookup[(bit_grid[L*((row+1)%L) + col] >> threadIdx.x) & one];
      n2 = llookup[(bit_grid[L*((row+L-1)%L) + col] >> threadIdx.x) & one];
      n3 = llookup[(bit_grid[L*row + (col+1)%L] >> threadIdx.x) & one];
      n4 = llookup[(bit_grid[L*row + (col+L-1)%L] >> threadIdx.x) & one];

      //n_sum = 4;
      index = 5*(spin+1) + n1 + n2 + n3 + n4 + 4;

      if (hiprand_uniform(&localState) < d_Pacc[index] ) {
          // accept - toggle bit
          bit_grid[my_idx] ^= one << threadIdx.x;
          //atomicXor(&bit_grid[my_idx],one << threadIdx.x);  // unsupported for uint64_t
      } 
      
      
    } //end for

    // Copy local data back to device global memory
    state[idx] = localState;

    for (row=0;row<L;row++){
      for (col=0;col<L;col++){
        my_idx = L*row + col;
        d_ising_grids[N*idx+my_idx] = llookup[(bit_grid[my_idx] >> threadIdx.x) & one];
      }
    }

  }

  return;

}

// compute magnetisation on the gpu
__global__ void compute_magnetisation_gpu(const int L, const int ngrids, int *d_ising_grids, float *d_magnetisation) {

  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  if (idx < ngrids) {

    int *loc_grid = &d_ising_grids[idx*L*L]; // pointer to device global memory

    float m = 0.0f;

    int i;
    for (i=0;i<L*L;i++) { m += loc_grid[i]; }
    d_magnetisation[idx] = m/(float)(L*L);

  }

  return;

}
